#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "Point.h"
#include "Camera.h"
#include <iostream>
#include <SDL.h>

__global__ void kernel(unsigned char* pixels, int* int_data, float* float_data, float* objects) {

    const int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    const int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    const int max_grid = int_data[0];
    const int max_block = int_data[1];
    const int texture_height = int_data[2];

    const int x = max_grid * blockIdx.y + blockIdx.x;
    const int y = max_block * threadIdx.y + threadIdx.x;

    float end_point_x = float_data[0] + float_data[3] * x;
    float end_point_y = float_data[1] + float_data[4] * y;
    float end_point_z = float_data[2] + float_data[5] * x;

    const float to_camera_x = end_point_x;
    const float to_camera_y = end_point_y;
    const float to_camera_z = end_point_z;

    float camera_x = float_data[6];
    float camera_y = float_data[7];
    float camera_z = float_data[8];

    // !!!
    const int number_of_objects = 1;

    float length;
    float coff;

    float min_distance;
    float distance;
    int idx;

    for (int i = 0; i < 20; i++) {
        min_distance = 0;
        distance = 0;
        idx = 0;
        for (int j = 0; j < 1; j++) {
            if (objects[j * 10] == 1) {
                distance = sqrtf(
                    powf(objects[j * 10 + 1] - camera_x, 2) +
                    powf(objects[j * 10 + 2] - camera_y, 2) +
                    powf(objects[j * 10 + 3] - camera_z, 2)); // - ������ �����
                distance -= objects[j * 10 + 7];
            }
            if (distance < min_distance || j == 0) {
                min_distance = distance;
                idx = j;
            }
            if (min_distance < 0.01) {
                if (objects[j * 10] == 1) {
                    pixels[(y * texture_height + x) * 4] = objects[idx * 10 + 6];
                    pixels[(y * texture_height + x) * 4 + 1] = objects[idx * 10 + 5];
                    pixels[(y * texture_height + x) * 4 + 2] = objects[idx * 10 + 4];
                    return;
                }
            }
            else if (min_distance > 20) {
                return;
            }
        }

        end_point_x = camera_x + to_camera_x;
        end_point_y = camera_y + to_camera_y;
        end_point_z = camera_z + to_camera_z;

        length = sqrtf(
            powf(end_point_x - camera_x, 2) +
            powf(end_point_y - camera_y, 2) +
            powf(end_point_z - camera_z, 2));
        coff = min_distance / length;

        camera_x = camera_x + (end_point_x - camera_x) * coff;
        camera_y = camera_y + (end_point_y - camera_y) * coff;
        camera_z = camera_z + (end_point_z - camera_z) * coff;
    }
}

int main(int argc, char** argv) {

    const unsigned int screen_width = 512;
    const unsigned int screen_height = 512;

    const unsigned int texture_width = 512;
    const unsigned int texture_height = 512;

    const int max_grid = ceil(sqrt(texture_width));
    const int max_block = ceil(sqrt(texture_height));

    dim3 threadsPerBlock(max_block, max_block);
    dim3 numBlocks(max_grid, max_grid);

    Point* camera_coordinates = new Point();
    Camera* camera = new Camera();

    SDL_Init(SDL_INIT_EVERYTHING);

    SDL_Window* window = SDL_CreateWindow
    (
        "SDL2",
        SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED,
        screen_width, screen_height,
        SDL_WINDOW_SHOWN
    );

    SDL_Renderer* renderer = SDL_CreateRenderer
    (
        window,
        -1,
        SDL_RENDERER_ACCELERATED
    );

    SDL_RendererInfo info;
    SDL_GetRendererInfo(renderer, &info);

    SDL_Texture* texture = SDL_CreateTexture
    (
        renderer,
        SDL_PIXELFORMAT_ARGB8888,
        SDL_TEXTUREACCESS_STREAMING,
        texture_width, texture_height
    );

    // int* pixels = new int[texture_width * texture_height * 4]{ 100 };
    // CUDA

    size_t size = (texture_width * texture_height * 4) * sizeof(unsigned char);
    size_t int_data_size = 3 * sizeof(int);
    size_t float_data_size = 10 * sizeof(float);
    size_t objects_size = 10 * 1 * sizeof(float);

    unsigned char* pixels = (unsigned char*)malloc(size);
    for (int i = 0; i < (texture_width * texture_height * 4); i++) {
        pixels[i] = 0;
    }

    int* int_data = (int*)malloc(int_data_size);
    int_data[0] = max_grid;
    int_data[1] = max_block;
    int_data[2] = texture_height;

    float* float_data = (float*)malloc(float_data_size);
    camera->update();
    float_data[0] = camera->get_start_x();
    float_data[1] = camera->get_start_y();
    float_data[2] = camera->get_start_z();
    float_data[3] = camera->get_delta_x() / texture_width;
    float_data[4] = camera->get_delta_y() / texture_height;
    float_data[5] = camera->get_delta_z() / texture_width;
    float_data[6] = camera_coordinates->get_x();
    float_data[7] = camera_coordinates->get_y();
    float_data[8] = camera_coordinates->get_z();

    float* objects = (float*)malloc(objects_size);
    objects[0] = 1;
    objects[1] = 5;
    objects[2] = 0;
    objects[3] = 0;
    objects[4] = 255;
    objects[5] = 0;
    objects[6] = 0;
    objects[7] = 3;
    objects[8] = 0;
    objects[9] = 0;

    unsigned char* d_pixels;
    int* d_int_data;
    float* d_float_data;
    float* d_objects;

    hipMalloc(&d_pixels, size);
    hipMalloc(&d_int_data, int_data_size);
    hipMalloc(&d_float_data, float_data_size);
    hipMalloc(&d_objects, objects_size);

    hipMemcpy(d_pixels, pixels, size, hipMemcpyHostToDevice);
    hipMemcpy(d_int_data, int_data, int_data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_float_data, float_data, float_data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_objects, objects, objects_size, hipMemcpyHostToDevice);

    SDL_Event event;
    bool running = true;

    unsigned int frames = 0;
    Uint64 start = SDL_GetPerformanceCounter();

    while (running) {
        SDL_SetRenderDrawColor(renderer, 0, 0, 0, SDL_ALPHA_OPAQUE);
        SDL_RenderClear(renderer);

        while (SDL_PollEvent(&event))
        {
            if ((SDL_QUIT == event.type) || (SDL_KEYDOWN == event.type && SDL_SCANCODE_ESCAPE == event.key.keysym.scancode))
            {
                running = false;
                break;
            }
        }

        hipMemcpy(d_int_data, int_data, int_data_size, hipMemcpyHostToDevice);
        hipMemcpy(d_float_data, float_data, float_data_size, hipMemcpyHostToDevice);
        hipMemcpy(d_objects, objects, objects_size, hipMemcpyHostToDevice);

        kernel <<<numBlocks, threadsPerBlock>>> (d_pixels, d_int_data, d_float_data, d_objects);

        hipMemcpy(pixels, d_pixels, size, hipMemcpyDeviceToHost);

        SDL_UpdateTexture
        (
            texture,
            NULL,
            pixels,
            texture_width * 4
        );

        SDL_RenderCopy(renderer, texture, NULL, NULL);
        SDL_RenderPresent(renderer);

        frames++;
        const Uint64 end = SDL_GetPerformanceCounter();
        const static Uint64 freq = SDL_GetPerformanceFrequency();
        const double seconds = (end - start) / static_cast<double>(freq);
        if (seconds > 2.0)
        {
            std::cout
                << frames << " frames in "
                << std::fixed << seconds << " seconds = "
                << std::fixed << frames / seconds << " FPS ("
                << std::fixed << (seconds * 1000.0) / frames << " ms/frame)"
                << std::endl;
            start = end;
            frames = 0;
        }
    }

    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    hipFree(d_pixels);
    hipFree(d_int_data);
    hipFree(d_float_data);

    return 0;
}
